#include "hip/hip_runtime.h"

#include <stdio.h>
#include "slenet_params.h"

#define INSIZE 28
#define INFO_BYTE_SIZE 4

#define INITIAL_WEIGHT_VALUE -1.0f
#define INITIAL_FC_WEIGHT_VALUE 1.0f
#define IMAGE_WIDTH 28
#define IMAGE_HEIGHT 28
#define CONV_FILTER 5
#define SS_FILTER 4
#define FEATURES 6 
#define NEURONS 10
#define CONV_OUTPUT 24
#define SS_OUTPUT 6
#define FC_OUTPUT 10

//kernel function that fill mnist_data structure->data with normalized pixel values
__global__ void fillArr(unsigned char pixels[INSIZE][INSIZE], double data[INSIZE][INSIZE]){
  // TO DO
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if(i<INSIZE && j<INSIZE) 
    data[i][j] = pixels[i][j]/255.0;
}

//kernel function that changes the values >0 to 1 and double type to integer type
__global__ void showArr(double ddata[INSIZE][INSIZE], int dshow[INSIZE][INSIZE]){
  // TO DO
  const int i = threadIdx.x + blockIdx.x * blockDim.x;
  const int j = threadIdx.y + blockIdx.y * blockDim.y;
  if(i<INSIZE && j<INSIZE){
    if(ddata[i][j]>0)
      dshow[i][j] = 1;
    else dshow[i][j] = 0;
  }
}

//mnist data structure
typedef struct mnist_data{
  double data[INSIZE][INSIZE];
  unsigned int label;
}mnist_data;

//structure for images header information
typedef struct images_info{
  char magic_num_images[INFO_BYTE_SIZE];
  char amount_images[INFO_BYTE_SIZE];
  char rows[INFO_BYTE_SIZE];
  char columns[INFO_BYTE_SIZE];
}images_info;

//structure for labels header information
typedef struct labels_info{
  char magic_num_labels[INFO_BYTE_SIZE];
  char amount_labels[INFO_BYTE_SIZE];
}labels_info;

//Hexadecimal to integer
static unsigned int mnist_bin_to_int(char *tmp){
    int val = (tmp[0] << 24 | tmp[1] << 16 | tmp[2] << 8 | tmp[3] );
    return val;
}

static int mnist_load(const char *image_filename, const char *label_filename, mnist_data **data_set,unsigned int *count){
                          
    images_info i_info;
    labels_info l_info;

    //opening the files
    FILE *images = fopen(image_filename,"rb");
    FILE *labels = fopen(label_filename,"rb");
    if(images==NULL||labels==NULL){
        return -1;
    }

    //read header info
    fread(&i_info,sizeof(images_info),1,images);
    fread(&l_info,sizeof(labels_info),1,labels);

    //check and print header info
    int magic_num_images_as_int = mnist_bin_to_int(i_info.magic_num_images);
    if(magic_num_images_as_int != 2051){
      printf("Problems with 'image magic number'. It is equal to %d, but should be 2051.",magic_num_images_as_int);
      return -1;
    }
    else{
      printf("image magic number = %d (should be 2051)\n", magic_num_images_as_int);   
    }

    int magic_num_labels_as_int = mnist_bin_to_int(l_info.magic_num_labels);
    if(magic_num_labels_as_int != 2049){
      printf("Problems with 'label magic number'. It is equal to %d, but should be 2049.",magic_num_labels_as_int);
      return -1;
    }
    else{
      printf("label magic number = %d (should be 2049)\n", magic_num_labels_as_int); 
    }

    int amount_images_as_int = mnist_bin_to_int(i_info.amount_images);
    if(amount_images_as_int != 10000){
      printf("Problems with 'image total number'. It is equal to %d, but should be 10000.",amount_images_as_int);
      return -1;
    }
    else{
      printf("image total number = %d (should be 10000)\n", amount_images_as_int); 
    }

    int amount_labels_as_int = mnist_bin_to_int(l_info.amount_labels);
    if(amount_labels_as_int != 10000){
      printf("Problems with 'label total number'. It is equal to %d, but should be 10000.",amount_labels_as_int);
      return -1;
    }
    else{
      printf("label total number = %d (should be 10000)\n", amount_labels_as_int);
    }

    int rows_as_int = mnist_bin_to_int(i_info.rows);
    int columns_as_int = mnist_bin_to_int(i_info.columns);
    if((rows_as_int != 28)||(columns_as_int!=28)){
      printf("Problems with dimensions of images. Dimensions of images are not compitable with 28x28.");
      return -1;
    }
    else{
      printf("rows = %d, cols = %d (both should be 28)\n", rows_as_int,columns_as_int);
    }

    unsigned char pixels[INSIZE][INSIZE];
    char label;

    for(int k = 0;k<10000;k++){
        
      //read current necessary data point
      fread(pixels,sizeof(pixels),1,images);
      fread(&label,sizeof(char),1,labels);


      //fill mnist_data struct -> data array with double values of pixels using cuda    
      unsigned char (*dpixels)[INSIZE];
      double (*ddata)[INSIZE];

      hipMalloc((void**)&dpixels, INSIZE*INSIZE*sizeof(char));
      hipMalloc((void**)&ddata, INSIZE*INSIZE*sizeof(double));

      hipMemcpy(dpixels, pixels, INSIZE*INSIZE*sizeof(unsigned char), hipMemcpyHostToDevice);

      dim3 blocks(1,1);
      dim3 threads(INSIZE,INSIZE);
      fillArr<<<blocks, threads>>>(dpixels,ddata);

      hipMemcpy((*data_set+*count)->data, ddata, INSIZE*INSIZE*sizeof(double), hipMemcpyDeviceToHost);

      hipFree(dpixels); 
      hipFree(ddata);

      //assign mnist_data struct -> label with label 
      (*data_set+*count)->label = (int)label;

      //increment count
      *count+=1;
    }

    //close files
    fclose(images);
    fclose(labels);

    return 0;
}

//Convolution layer. Filtering.
__global__ void conv_filtering(float d_data[28][28],
                    float d_weight[6][5][5],
                    float d_pre_output[6][24][24]){
        
  const int local_row = threadIdx.y;
  const int local_column = threadIdx.z;
  const int feature = threadIdx.x;
  const int global_row = blockIdx.x+threadIdx.y;
  const int global_column = blockIdx.y+threadIdx.z;
  const int output_row = blockIdx.x;
  const int output_column = blockIdx.y;
  __shared__ float temp[FEATURES][CONV_FILTER][CONV_FILTER];
  __shared__ float pre_sum[FEATURES][CONV_FILTER];
  temp[feature][local_row][local_column] = d_data[global_row][global_column]*d_weight[feature][local_row][local_column];
  __syncthreads();
  if(local_column==0){
      float temp_sum = 0.0f;
      for(int i =0; i< CONV_FILTER;i++){
       temp_sum+=temp[feature][local_row][i];
      }
      pre_sum[feature][local_row] = temp_sum;
      __syncthreads();
      if(local_row==0){
        float sum = 0.0f;
        for(int i =0; i< CONV_FILTER;i++){
          sum+=pre_sum[feature][i];
        }
        d_pre_output[feature][output_row][output_column] = sum;
      }
  }
}

//Convolution layer. Biasing.
__global__ void conv_biasing(float d_pre_output[6][24][24],
                    float d_bias[6]){
                        
    const int x = blockIdx.x*blockDim.x+threadIdx.x;
    const int y = blockIdx.y*blockDim.y+threadIdx.y;
    const int feature = blockIdx.z;

    d_pre_output[feature][x][y] += d_bias[feature];
}

//Convolution layer. Sigmoid.
__global__ void conv_sigmoid(float d_pre_output[6][24][24],
                    float d_output[6][24][24]){
                        
    const int x = blockIdx.x*blockDim.x+threadIdx.x;
    const int y = blockIdx.y*blockDim.y+threadIdx.y;
    const int feature = blockIdx.z;

    d_output[feature][x][y] = 1/(1+expf((-1)*d_pre_output[feature][x][y]));
}

//SubSampling layer. Filtering.
__global__ void ss_filtering(float d_conv_output[6][24][24],
                    float d_weight[4][4],
                    float d_pre_output[6][6][6]){
        
  const int local_row = threadIdx.y;
  const int local_column = threadIdx.z;
  const int feature = threadIdx.x;
  const int global_row = blockIdx.x*blockDim.y+threadIdx.y;
  const int global_column = blockIdx.y*blockDim.z+threadIdx.z;
  const int output_row = blockIdx.x;
  const int output_column = blockIdx.y;
  __shared__ float temp[FEATURES][SS_FILTER][SS_FILTER];
  temp[feature][local_row][local_column] = d_conv_output[feature][global_row][global_column]*d_weight[local_row][local_column];
  __syncthreads();
  if(local_row==0 && local_column==0){
      float sum = 0.0f;
      for(int i = 0; i<SS_FILTER; i++){
          for(int j =0; j<SS_FILTER; j++){
              sum+=temp[feature][i][j];
          }
      }
      d_pre_output[feature][output_row][output_column] = sum;
  }
}

//SubSampling layer. Biasing.
__global__ void ss_biasing(float d_pre_output[6][6][6],
                    float d_bias[1]){
                        
    const int x = threadIdx.x;
    const int y = threadIdx.y;
    const int feature = blockIdx.x;

    d_pre_output[feature][x][y] += d_bias[0];
}

//SubSampling layer. Sigmoid.
__global__ void ss_sigmoid(float d_pre_output[6][6][6],
                    float d_output[6][6][6]){
                        
    const int x = threadIdx.x;
    const int y = threadIdx.y;
    const int feature = blockIdx.x;

    d_output[feature][x][y] = 1/(1+expf((-1)*d_pre_output[feature][x][y]));
}

__global__ void fc_linear(float d_ss_output[6][6][6], float d_weight[10][6][6][6],float d_pre_output[10]){
    const int neuron = blockIdx.x;
    const int depth = blockIdx.y*blockDim.x+threadIdx.x;
    const int local_depth = threadIdx.x;
    const int row = threadIdx.y;
    const int column = threadIdx.z;
    __shared__ float temp[3][6][6];
    __shared__ float temp_sums[3][6];
    __shared__ float pre_sums[3];
    temp[local_depth][row][column] = d_ss_output[depth][row][column]*d_weight[neuron][depth][row][column];
    __syncthreads();
    if(column==0){
      float temp_sum = 0.0f;
      for(int i = 0; i<6;i++){
        temp_sum+=temp[local_depth][row][i];  
      }
      temp_sums[local_depth][row] = temp_sum;
      if(row==0){
        float pre_sum = 0.0f;
        for(int i = 0; i<6;i++){
          pre_sum+=temp_sums[local_depth][i];  
        }
        pre_sums[local_depth] = pre_sum;
        if(local_depth==0){
          float sum = 0.0f;
          for(int i = 0; i<3;i++){
            sum+=pre_sums[i];  
          }
          atomicAdd(&d_pre_output[neuron],sum);
        }
      }
    }
}

//Fully-connected layer.Biasing.
__global__ void fc_biasing(float d_pre_output[10], float d_bias[10]){
    const int idx = threadIdx.x;
    d_pre_output[idx] += d_bias[idx];
}

//Fully-connected layer.Sigmoid.
__global__ void fc_sigmoid(float d_pre_output[10], float d_output[10]){
    const int idx = threadIdx.x;
    d_output[idx] = 1/(1+expf((-1)*d_pre_output[idx]));
}

class Conv{
  public:
    int filter_size, features_num, output_dim;
    float *weight, *bias,*pre_output, *output;
    Conv(int filter_size, int features_num, int output);
    void reset();
    ~Conv();

};

Conv::Conv(int filter_size, int features_num, int output_dim){
    
    //Assigning attributes
    this->filter_size = filter_size;
    this->features_num = features_num;
    this->output_dim = output_dim;

    //CUDA memory allocation
    hipMalloc((void **)&weight, features_num*filter_size*filter_size*sizeof(float));
    hipMemcpy(weight, c1_weight, features_num*filter_size*filter_size*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&bias, features_num*sizeof(float));
    hipMemcpy(bias, c1_bias, features_num*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&pre_output, features_num*output_dim*output_dim*sizeof(float));
    hipMalloc((void **)&output, features_num*output_dim*output_dim*sizeof(float));
}

void Conv::reset(){
    hipMemset(pre_output,0x00, features_num*output_dim*output_dim*sizeof(float));
    hipMemset(output,0x00,features_num*output_dim*output_dim*sizeof(float));
}

Conv::~Conv(){
    
  //CUDA memory deallocation  
  hipFree(weight);
  hipFree(bias);
  hipFree(pre_output);
  hipFree(output);
}

class SS{
  public:
    int filter_size, features_num, output_dim;
    float *weight, *bias,*pre_output, *output;
    SS(int filter_size, int features_num, int output);
    void reset();
    ~SS();
};

SS::SS(int filter_size, int features_num, int output_dim){
    
    //Assigning attributes
    this->filter_size = filter_size;
    this->features_num = features_num;
    this->output_dim = output_dim;

    //CUDA memory allocation
    hipMalloc((void **)&weight, filter_size*filter_size*sizeof(float));
    hipMemcpy(weight, s2_weight, filter_size*filter_size*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void **)&bias, filter_size*filter_size*sizeof(float));
    hipMemcpy(bias, s2_bias, sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&pre_output, features_num*output_dim*output_dim*sizeof(float));
    hipMalloc((void **)&output, features_num*output_dim*output_dim*sizeof(float));
}

void SS::reset(){
    hipMemset(pre_output,0x00, features_num*output_dim*output_dim*sizeof(float));
    hipMemset(output,0x00,features_num*output_dim*output_dim*sizeof(float));
}


SS::~SS(){
    
  //CUDA memory deallocation  
  hipFree(weight);
  hipFree(bias);
  hipFree(pre_output);
  hipFree(output);
}

class FC{
  public:
    int  neurons, output_dim;
    float *weight, *bias,*pre_output, *output;
    FC(int neurons, int output);
    void reset();
    ~FC();

};

FC::FC(int neurons, int output_dim){
    
    //Assigning attributes
    this->neurons = neurons;
    this->output_dim = output_dim;

    //CUDA memory allocation
    hipMalloc((void **)&weight, neurons*FEATURES*SS_OUTPUT*SS_OUTPUT*sizeof(float));
    hipMemcpy(weight, f3_weight, neurons*FEATURES*SS_OUTPUT*SS_OUTPUT*sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void **)&bias, neurons*sizeof(float));
    hipMemcpy(bias, f3_bias, neurons*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&pre_output, output_dim*sizeof(float));
    hipMalloc((void **)&output, output_dim*sizeof(float));
}
void FC::reset(){
    hipMemset(pre_output,0x00, output_dim*sizeof(float));
    hipMemset(output,0x00,output_dim*sizeof(float));
}

FC::~FC(){
    
  //CUDA memory deallocation  
  hipFree(weight);
  hipFree(bias);
  hipFree(pre_output);
  hipFree(output);
}

static Conv conv = Conv(CONV_FILTER, FEATURES, CONV_OUTPUT);
static SS ss = SS(SS_FILTER, FEATURES, SS_OUTPUT);
static FC fc = FC(NEURONS, FC_OUTPUT);

//Forward pass 
static float forward_pass(float data[IMAGE_WIDTH][IMAGE_HEIGHT]){ //unsigned int label, unsigned int *error){
    
    conv.reset();
    hipError_t conv_reset_checker = hipGetLastError();
    if (conv_reset_checker!=hipSuccess){
      printf("CONV reset PROBLEM:: %s", hipGetErrorString(conv_reset_checker));
      exit(1);
    }
    ss.reset();
    hipError_t ss_reset_checker = hipGetLastError();
    if (ss_reset_checker!=hipSuccess){
      printf("ss reset PROBLEM:: %s", hipGetErrorString(ss_reset_checker));
      exit(1);
    }
    fc.reset();
    hipError_t fc_reset_checker = hipGetLastError();
    if (fc_reset_checker!=hipSuccess){
      printf("fc reset PROBLEM:: %s", hipGetErrorString(fc_reset_checker));
      exit(1);
    }
    
    float (*kernel_data)[IMAGE_HEIGHT];

    float time = 0.0f;
    float ms = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((void**)&kernel_data,IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(float));
    hipMemcpy(kernel_data, data, IMAGE_WIDTH*IMAGE_HEIGHT*sizeof(float), hipMemcpyHostToDevice);

    dim3 conv_filter_blocks(CONV_OUTPUT, CONV_OUTPUT);
    dim3 conv_filter_thread(FEATURES, CONV_FILTER, CONV_FILTER);
    hipEventRecord(start);
    conv_filtering<<<conv_filter_blocks, conv_filter_thread>>>(kernel_data,
                                         (float (*)[CONV_FILTER][CONV_FILTER])conv.weight,
                                         (float (*)[CONV_OUTPUT][CONV_OUTPUT])conv.pre_output);
    hipError_t conv_filter_checker = hipGetLastError();
    if (conv_filter_checker!=hipSuccess){
      printf("CONV FILTERING PROBLEM:: %s", hipGetErrorString(conv_filter_checker));
      exit(1);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    time+=ms;

    int conv_block_dim = CONV_OUTPUT/3;
    dim3 conv_bias_blocks(CONV_OUTPUT/conv_block_dim,CONV_OUTPUT/conv_block_dim,FEATURES);
    dim3 conv_bias_thread(conv_block_dim,conv_block_dim);
    hipEventRecord(start);
    conv_biasing<<<conv_bias_blocks, conv_bias_thread>>>((float (*)[CONV_OUTPUT][CONV_OUTPUT])conv.pre_output, 
                                                         conv.bias);
    hipError_t conv_bias_checker = hipGetLastError();
    if (conv_bias_checker!=hipSuccess){
      printf("CONV BIASING PROBLEM:: %s", hipGetErrorString(conv_bias_checker));
      exit(1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    time+=ms;
    
    dim3 conv_sigmoid_blocks(CONV_OUTPUT/conv_block_dim,CONV_OUTPUT/conv_block_dim,FEATURES);
    dim3 conv_sigmoid_thread(conv_block_dim,conv_block_dim);
    hipEventRecord(start);
    conv_sigmoid<<<conv_sigmoid_blocks, conv_sigmoid_thread>>>((float (*)[CONV_OUTPUT][CONV_OUTPUT])conv.pre_output,
                                                    (float (*)[CONV_OUTPUT][CONV_OUTPUT])conv.output);
    hipError_t conv_sigmoid_checker = hipGetLastError();
    if (conv_sigmoid_checker!=hipSuccess){
      printf("CONV SIGMOID PROBLEM:: %s", hipGetErrorString(conv_sigmoid_checker));
      exit(1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    time+=ms;

    dim3 ss_filter_blocks(SS_OUTPUT, SS_OUTPUT);
    dim3 ss_filter_thread(FEATURES, SS_FILTER, SS_FILTER);
    hipEventRecord(start);
    ss_filtering<<<ss_filter_blocks, ss_filter_thread>>>((float (*)[CONV_OUTPUT][CONV_OUTPUT])conv.output,
                                         (float (*)[SS_FILTER])ss.weight,
                                         (float (*)[SS_OUTPUT][SS_OUTPUT])ss.pre_output);
    hipError_t ss_filter_checker = hipGetLastError();
    if (ss_filter_checker!=hipSuccess){
      printf("SS FILTERING PROBLEM:: %s", hipGetErrorString(ss_filter_checker));
      exit(1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    time+=ms;
      
    dim3 ss_bias_blocks(FEATURES);
    dim3 ss_bias_thread(SS_OUTPUT,SS_OUTPUT);
    hipEventRecord(start);
    ss_biasing<<<ss_bias_blocks, ss_bias_thread>>>((float (*)[SS_OUTPUT][SS_OUTPUT])ss.pre_output, (float (*))ss.bias);
    hipError_t ss_bias_checker = hipGetLastError();
    if (ss_bias_checker!=hipSuccess){
      printf("SS BIASING PROBLEM:: %s", hipGetErrorString(ss_bias_checker));
      exit(1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    time+=ms;
    
    dim3 ss_sigmoid_blocks(FEATURES);
    dim3 ss_sigmoid_thread(SS_OUTPUT,SS_OUTPUT);
    hipEventRecord(start);
    ss_sigmoid<<<ss_sigmoid_blocks, ss_sigmoid_thread>>>((float (*)[SS_OUTPUT][SS_OUTPUT])ss.pre_output,
                                                    (float (*)[SS_OUTPUT][SS_OUTPUT])ss.output);
    hipError_t ss_sigmoid_checker = hipGetLastError();
    if (ss_sigmoid_checker!=hipSuccess){
      printf("SS SIGMOID PROBLEM:: %s", hipGetErrorString(ss_sigmoid_checker));
      exit(1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    time+=ms;

    int div = FEATURES/2;
    dim3 fc_linear_blocks(FC_OUTPUT, FEATURES/div);
    dim3 fc_linear_thread(div, SS_OUTPUT, SS_OUTPUT);
    hipEventRecord(start);
    fc_linear<<<fc_linear_blocks, fc_linear_thread>>>((float (*)[SS_OUTPUT][SS_OUTPUT])ss.output,
                                         (float (*)[FEATURES][SS_OUTPUT][SS_OUTPUT])fc.weight,
                                         fc.pre_output);
    hipError_t fc_linear_checker = hipGetLastError();
    if (fc_linear_checker!=hipSuccess){
      printf("FC LINEAR PROBLEM:: %s", hipGetErrorString(fc_linear_checker));
      exit(1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    time+=ms;
      
    dim3 fc_bias_blocks(1);
    dim3 fc_bias_thread(NEURONS);
    hipEventRecord(start);
    fc_biasing<<<fc_bias_blocks, fc_bias_thread>>>(fc.pre_output, fc.bias);
    hipError_t fc_bias_checker = hipGetLastError();
    if (fc_bias_checker!=hipSuccess){
      printf("FC BIASING PROBLEM:: %s", hipGetErrorString(fc_bias_checker));
      exit(1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    time+=ms;
    
    dim3 fc_sigmoid_blocks(1);
    dim3 fc_sigmoid_thread(NEURONS);
    hipEventRecord(start);
    fc_sigmoid<<<fc_sigmoid_blocks, fc_sigmoid_thread>>>(fc.pre_output,fc.output);
    hipError_t fc_sigmoid_checker = hipGetLastError();
    if (fc_sigmoid_checker!=hipSuccess){
      printf("FC SIGMOID PROBLEM:: %s", hipGetErrorString(fc_sigmoid_checker));
      exit(1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    time+=ms;
    
    hipFree(kernel_data);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    return time;
}

int main(){
    
    const char *image_filename = "data/t10k-images.idx3-ubyte";
    const char *label_filename = "data/t10k-labels.idx1-ubyte";
    mnist_data *data_set = (mnist_data *)malloc(sizeof(*data_set)*10000);
    unsigned int count = 0;
    
    if(mnist_load(image_filename,label_filename, &data_set,&count)!=0){
      printf("Problems with loading data.");
      exit(1);
    }
    printf("test_cnt = %d (should be 10000)\n\n",count);
    
    unsigned int error = 0;
    float time_taken = 0.0f;
    for(int k = 0; k<count;k++){

      float data[IMAGE_HEIGHT][IMAGE_WIDTH];
      for(int i = 0; i< IMAGE_HEIGHT;i++){
          for(int j = 0; j< IMAGE_WIDTH;j++){
              data[i][j] = data_set[k].data[i][j];
          }
      }
      time_taken += forward_pass(data);
      unsigned int max = 0;
      float res[10];
      hipMemcpy(res, fc.output, sizeof(float)*10, hipMemcpyDeviceToHost);
      for(int j=0; j<10; j++){
        if (res[max] < res[j])
          max = j;
      }
      if(max!=data_set[k].label) error+=1;
    }
    printf("Error Rate = %f%% (%d out of 10,000)\n", double(error)/double(count)*100.0, error);
    printf("Accuracy = %.3f%% (%d out of 10,000)\n", 100.0 - double(error)/double(count)*100.0, count - error);
    printf("Ex time = %f (ms) \n", time_taken);

    return 0;
}
